#include "hip/hip_runtime.h"
//#include "sys/time.h"
#define WIN32 // SO STUPID WTF

#include <iostream>
#include <functional>
#include <string>

#include "time.h"
#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime_api.h"
#include "helper_functions.h"

#include <stdio.h>
#include <time.h>

#include "kernel.h"
#include "utils_cpu.h"


/* F, G and H are basic MD5 functions: selection, majority, parity */
#define F(x, y, z) (((x) & (y)) | ((~x) & (z)))
#define G(x, y, z) (((x) & (z)) | ((y) & (~z)))
#define H(x, y, z) ((x) ^ (y) ^ (z))
#define I(x, y, z) ((y) ^ ((x) | (~z))) 

/* ROTATE_LEFT rotates x left n bits */
#define ROTATE_LEFT(x, n) (((x) << (n)) | ((x) >> (32-(n))))

/* FF, GG, HH, and II transformations for rounds 1, 2, 3, and 4 */
/* Rotation is separate from addition to prevent recomputation */
#define FF(a, b, c, d, x, s, ac) \
  {(a) += F ((b), (c), (d)) + (x) + (unsigned int)(ac); \
   (a) = ROTATE_LEFT ((a), (s)); \
   (a) += (b); \
  }
#define GG(a, b, c, d, x, s, ac) \
  {(a) += G ((b), (c), (d)) + (x) + (unsigned int)(ac); \
   (a) = ROTATE_LEFT ((a), (s)); \
   (a) += (b); \
  }
#define HH(a, b, c, d, x, s, ac) \
  {(a) += H ((b), (c), (d)) + (x) + (unsigned int)(ac); \
   (a) = ROTATE_LEFT ((a), (s)); \
   (a) += (b); \
  }
#define II(a, b, c, d, x, s, ac) \
  {(a) += I ((b), (c), (d)) + (x) + (unsigned int)(ac); \
   (a) = ROTATE_LEFT ((a), (s)); \
   (a) += (b); \
  }

// Accessor for w[16] array. Naively, this would just be w[i]; however, this
// choice leads to worst-case-scenario access pattern wrt. shared memory
// bank conflicts, as the same indices in different threads fall into the
// same bank (as the words are 16 unsigned ints long). The packing below causes the
// same indices in different threads of a warp to map to different banks. In
// testing this gave a ~40% speedup.
//
// PS: An alternative solution would be to make the w array 17 unsigned ints long
// (thus wasting a little shared memory)
//
__device__ inline unsigned int &getw(unsigned int *w, const int i)
{
	return w[(i+threadIdx.x) % 16];
}

__device__ inline unsigned int getw(const unsigned int *w, const int i)	// const- version
{
	return w[(i+threadIdx.x) % 16];
}



void inline __device__ GPUshufflegetw(unsigned int* in)
{
  unsigned int tmp[16];

  for(int i = 0; i < 16; i++)
    tmp[i] = in[i];

  for(int i = 0; i < 16; i++)
    getw(in, i) = tmp[i];
}

/* Basic MD5 step. Transform buf based on in.
 */
void inline __device__ md5_v2(const unsigned int *in, unsigned int &a, unsigned int &b, unsigned int &c, unsigned int &d)
{
  #define S11 7
  #define S12 12
  #define S13 17
  #define S14 22
  #define S21 5
  #define S22 9
  #define S23 14
  #define S24 20
  #define S31 4
  #define S32 11
  #define S33 16
  #define S34 23
  #define S41 6
  #define S42 10
  #define S43 15
  #define S44 21

	const unsigned int a0 = 0x67452301;
	const unsigned int b0 = 0xEFCDAB89;
	const unsigned int c0 = 0x98BADCFE;
	const unsigned int d0 = 0x10325476;

	//Initialize hash value for this chunk:
	a = a0;
	b = b0;
	c = c0;
	d = d0;
 
  /* Round 1 */
#ifdef GETW_OPT
  FF ( a, b, c, d, getw(in,  0), S11, 3614090360); /* 1 */
  FF ( d, a, b, c, getw(in,  1), S12, 3905402710); /* 2 */
  FF ( c, d, a, b, getw(in,  2), S13,  606105819); /* 3 */
  FF ( b, c, d, a, getw(in,  3), S14, 3250441966); /* 4 */
  FF ( a, b, c, d, getw(in,  4), S11, 4118548399); /* 5 */
  FF ( d, a, b, c, getw(in,  5), S12, 1200080426); /* 6 */
  FF ( c, d, a, b, getw(in,  6), S13, 2821735955); /* 7 */
  FF ( b, c, d, a, getw(in,  7), S14, 4249261313); /* 8 */
  FF ( a, b, c, d, getw(in,  8), S11, 1770035416); /* 9 */
  FF ( d, a, b, c, getw(in,  9), S12, 2336552879); /* 10 */
  FF ( c, d, a, b, getw(in, 10), S13, 4294925233); /* 11 */
  FF ( b, c, d, a, getw(in, 11), S14, 2304563134); /* 12 */
  FF ( a, b, c, d, getw(in, 12), S11, 1804603682); /* 13 */
  FF ( d, a, b, c, getw(in, 13), S12, 4254626195); /* 14 */
  FF ( c, d, a, b, getw(in, 14), S13, 2792965006); /* 15 */
  FF ( b, c, d, a, getw(in, 15), S14, 1236535329); /* 16 */
#else
  FF (a, b, c, d, in[ 0], S11, 0xd76aa478); /* 1 */
  FF (d, a, b, c, in[ 1], S12, 0xe8c7b756); /* 2 */
  FF (c, d, a, b, in[ 2], S13, 0x242070db); /* 3 */
  FF (b, c, d, a, in[ 3], S14, 0xc1bdceee); /* 4 */
  FF (a, b, c, d, in[ 4], S11, 0xf57c0faf); /* 5 */
  FF (d, a, b, c, in[ 5], S12, 0x4787c62a); /* 6 */
  FF (c, d, a, b, in[ 6], S13, 0xa8304613); /* 7 */
  FF (b, c, d, a, in[ 7], S14, 0xfd469501); /* 8 */
  FF (a, b, c, d, in[ 8], S11, 0x698098d8); /* 9 */
  FF (d, a, b, c, in[ 9], S12, 0x8b44f7af); /* 10 */
  FF (c, d, a, b, in[10], S13, 0xffff5bb1); /* 11 */
  FF (b, c, d, a, in[11], S14, 0x895cd7be); /* 12 */
  FF (a, b, c, d, in[12], S11, 0x6b901122); /* 13 */
  FF (d, a, b, c, in[13], S12, 0xfd987193); /* 14 */
  FF (c, d, a, b, in[14], S13, 0xa679438e); /* 15 */
  FF (b, c, d, a, in[15], S14, 0x49b40821); /* 16 */
#endif

 /* Round 2 */
#ifdef GETW_OPT
  GG ( a, b, c, d, getw(in,  1), S21, 4129170786); /* 17 */
  GG ( d, a, b, c, getw(in,  6), S22, 3225465664); /* 18 */
  GG ( c, d, a, b, getw(in, 11), S23,  643717713); /* 19 */
  GG ( b, c, d, a, getw(in,  0), S24, 3921069994); /* 20 */
  GG ( a, b, c, d, getw(in,  5), S21, 3593408605); /* 21 */
  GG ( d, a, b, c, getw(in, 10), S22,   38016083); /* 22 */
  GG ( c, d, a, b, getw(in, 15), S23, 3634488961); /* 23 */
  GG ( b, c, d, a, getw(in,  4), S24, 3889429448); /* 24 */
  GG ( a, b, c, d, getw(in,  9), S21,  568446438); /* 25 */
  GG ( d, a, b, c, getw(in, 14), S22, 3275163606); /* 26 */
  GG ( c, d, a, b, getw(in,  3), S23, 4107603335); /* 27 */
  GG ( b, c, d, a, getw(in,  8), S24, 1163531501); /* 28 */
  GG ( a, b, c, d, getw(in, 13), S21, 2850285829); /* 29 */
  GG ( d, a, b, c, getw(in,  2), S22, 4243563512); /* 30 */
  GG ( c, d, a, b, getw(in,  7), S23, 1735328473); /* 31 */
  GG ( b, c, d, a, getw(in, 12), S24, 2368359562); /* 32 */
#else
  GG (a, b, c, d, in[ 1], S21, 0xf61e2562); /* 17 */
  GG (d, a, b, c, in[ 6], S22, 0xc040b340); /* 18 */
  GG (c, d, a, b, in[11], S23, 0x265e5a51); /* 19 */
  GG (b, c, d, a, in[ 0], S24, 0xe9b6c7aa); /* 20 */
  GG (a, b, c, d, in[ 5], S21, 0xd62f105d); /* 21 */
  GG (d, a, b, c, in[10], S22,  0x2441453); /* 22 */
  GG (c, d, a, b, in[15], S23, 0xd8a1e681); /* 23 */
  GG (b, c, d, a, in[ 4], S24, 0xe7d3fbc8); /* 24 */
  GG (a, b, c, d, in[ 9], S21, 0x21e1cde6); /* 25 */
  GG (d, a, b, c, in[14], S22, 0xc33707d6); /* 26 */
  GG (c, d, a, b, in[ 3], S23, 0xf4d50d87); /* 27 */
  GG (b, c, d, a, in[ 8], S24, 0x455a14ed); /* 28 */
  GG (a, b, c, d, in[13], S21, 0xa9e3e905); /* 29 */
  GG (d, a, b, c, in[ 2], S22, 0xfcefa3f8); /* 30 */
  GG (c, d, a, b, in[ 7], S23, 0x676f02d9); /* 31 */
  GG (b, c, d, a, in[12], S24, 0x8d2a4c8a); /* 32 */
#endif

  /* Round 3 */
#ifdef GETW_OPT
  HH ( a, b, c, d, getw(in,  5), S31, 4294588738); /* 33 */
  HH ( d, a, b, c, getw(in,  8), S32, 2272392833); /* 34 */
  HH ( c, d, a, b, getw(in, 11), S33, 1839030562); /* 35 */
  HH ( b, c, d, a, getw(in, 14), S34, 4259657740); /* 36 */
  HH ( a, b, c, d, getw(in,  1), S31, 2763975236); /* 37 */
  HH ( d, a, b, c, getw(in,  4), S32, 1272893353); /* 38 */
  HH ( c, d, a, b, getw(in,  7), S33, 4139469664); /* 39 */
  HH ( b, c, d, a, getw(in, 10), S34, 3200236656); /* 40 */
  HH ( a, b, c, d, getw(in, 13), S31,  681279174); /* 41 */
  HH ( d, a, b, c, getw(in,  0), S32, 3936430074); /* 42 */
  HH ( c, d, a, b, getw(in,  3), S33, 3572445317); /* 43 */
  HH ( b, c, d, a, getw(in,  6), S34,   76029189); /* 44 */
  HH ( a, b, c, d, getw(in,  9), S31, 3654602809); /* 45 */
  HH ( d, a, b, c, getw(in, 12), S32, 3873151461); /* 46 */
  HH ( c, d, a, b, getw(in, 15), S33,  530742520); /* 47 */
  HH ( b, c, d, a, getw(in,  2), S34, 3299628645); /* 48 */
#else
  HH (a, b, c, d, in[ 5], S31, 0xfffa3942); /* 33 */
  HH (d, a, b, c, in[ 8], S32, 0x8771f681); /* 34 */
  HH (c, d, a, b, in[11], S33, 0x6d9d6122); /* 35 */
  HH (b, c, d, a, in[14], S34, 0xfde5380c); /* 36 */
  HH (a, b, c, d, in[ 1], S31, 0xa4beea44); /* 37 */
  HH (d, a, b, c, in[ 4], S32, 0x4bdecfa9); /* 38 */
  HH (c, d, a, b, in[ 7], S33, 0xf6bb4b60); /* 39 */
  HH (b, c, d, a, in[10], S34, 0xbebfbc70); /* 40 */
  HH (a, b, c, d, in[13], S31, 0x289b7ec6); /* 41 */
  HH (d, a, b, c, in[ 0], S32, 0xeaa127fa); /* 42 */
  HH (c, d, a, b, in[ 3], S33, 0xd4ef3085); /* 43 */
  HH (b, c, d, a, in[ 6], S34,  0x4881d05); /* 44 */
  HH (a, b, c, d, in[ 9], S31, 0xd9d4d039); /* 45 */
  HH (d, a, b, c, in[12], S32, 0xe6db99e5); /* 46 */
  HH (c, d, a, b, in[15], S33, 0x1fa27cf8); /* 47 */
  HH (b, c, d, a, in[ 2], S34, 0xc4ac5665); /* 48 */
#endif

  /* Round 4 */
#ifdef GETW_OPT
  II ( a, b, c, d, getw(in,  0), S41, 4096336452); /* 49 */
  II ( d, a, b, c, getw(in,  7), S42, 1126891415); /* 50 */
  II ( c, d, a, b, getw(in, 14), S43, 2878612391); /* 51 */
  II ( b, c, d, a, getw(in,  5), S44, 4237533241); /* 52 */
  II ( a, b, c, d, getw(in, 12), S41, 1700485571); /* 53 */
  II ( d, a, b, c, getw(in,  3), S42, 2399980690); /* 54 */
  II ( c, d, a, b, getw(in, 10), S43, 4293915773); /* 55 */
  II ( b, c, d, a, getw(in,  1), S44, 2240044497); /* 56 */
  II ( a, b, c, d, getw(in,  8), S41, 1873313359); /* 57 */
  II ( d, a, b, c, getw(in, 15), S42, 4264355552); /* 58 */
  II ( c, d, a, b, getw(in,  6), S43, 2734768916); /* 59 */
  II ( b, c, d, a, getw(in, 13), S44, 1309151649); /* 60 */
  II ( a, b, c, d, getw(in,  4), S41, 4149444226); /* 61 */
  II ( d, a, b, c, getw(in, 11), S42, 3174756917); /* 62 */
  II ( c, d, a, b, getw(in,  2), S43,  718787259); /* 63 */
  II ( b, c, d, a, getw(in,  9), S44, 3951481745); /* 64 */
#else
  II (a, b, c, d, in[ 0], S41, 0xf4292244); /* 49 */
  II (d, a, b, c, in[ 7], S42, 0x432aff97); /* 50 */
  II (c, d, a, b, in[14], S43, 0xab9423a7); /* 51 */
  II (b, c, d, a, in[ 5], S44, 0xfc93a039); /* 52 */
  II (a, b, c, d, in[12], S41, 0x655b59c3); /* 53 */
  II (d, a, b, c, in[ 3], S42, 0x8f0ccc92); /* 54 */
  II (c, d, a, b, in[10], S43, 0xffeff47d); /* 55 */
  II (b, c, d, a, in[ 1], S44, 0x85845dd1); /* 56 */
  II (a, b, c, d, in[ 8], S41, 0x6fa87e4f); /* 57 */
  II (d, a, b, c, in[15], S42, 0xfe2ce6e0); /* 58 */
  II (c, d, a, b, in[ 6], S43, 0xa3014314); /* 59 */
  II (b, c, d, a, in[13], S44, 0x4e0811a1); /* 60 */
  II (a, b, c, d, in[ 4], S41, 0xf7537e82); /* 61 */
  II (d, a, b, c, in[11], S42, 0xbd3af235); /* 62 */
  II (c, d, a, b, in[ 2], S43, 0x2ad7d2bb); /* 63 */
  II (b, c, d, a, in[ 9], S44, 0xeb86d391); /* 64 */
#endif

	a += a0;
	b += b0;
	c += c0;
	d += d0;
}

#define BYTETOBINARYPATTERN "%d%d%d%d%d%d%d%d"
#define BYTETOBINARY(byte)  \
  (byte & 0x80 ? 1 : 0), \
  (byte & 0x40 ? 1 : 0), \
  (byte & 0x20 ? 1 : 0), \
  (byte & 0x10 ? 1 : 0), \
  (byte & 0x08 ? 1 : 0), \
  (byte & 0x04 ? 1 : 0), \
  (byte & 0x02 ? 1 : 0), \
  (byte & 0x01 ? 1 : 0) 

/* This code has an obvious bug and another non-obvious one :) */
__device__ inline void StoI(char *string, int length) {
  for(int i = 0; i < length; i++) {
    printf(BYTETOBINARYPATTERN" ", BYTETOBINARY(string[i]));
    if ((i+1) % 8 == 0)
      printf("\n");
  }
  printf("\n");
}


__device__ inline void Decode(unsigned int *output, unsigned char *input, unsigned int len) {
  unsigned int i, j;

  for (i = 0, j = 0; j < len; i++, j += 4) {
   output[i] = ((unsigned int)input[j]) | (((unsigned int)input[j+1]) << 8) |
     (((unsigned int)input[j+2]) << 16) | (((unsigned int)input[j+3]) << 24);
  }
}

__device__ inline void Encode(unsigned char *output, unsigned int *input, unsigned int len) {
  /* Encode()
   * converts a unsigned int[4] array into a uchar[16] array
  */
  unsigned int i, j;

  for (i = 0, j = 0; j < len; i++, j += 4) {
    output[j] = (unsigned char)(input[i] & 0xff);
    output[j+1] = (unsigned char)((input[i] >> 8) & 0xff);
    output[j+2] = (unsigned char)((input[i] >> 16) & 0xff);
    output[j+3] = (unsigned char)((input[i] >> 24) & 0xff);
  }
}



__device__ bool generatePermStarting(char *c0, int *starting, int pw_length, char* charset, int charset_len, int idx) {
  // modifies c0 to be a random string perm corresponding to idx in base(charset_len)
  int i;
  int idx_new;
  int idx_tmp;
  bool reset = true;

 for(i = 0; i < 64; i++) {
    if(i >= pw_length) {

      c0[i] = 0;

    } else {
      idx_tmp = idx+starting[i];
      if(idx_tmp < charset_len)
        reset = false;


      // old
      //c0[i] = charset[(idx+starting[i]) % charset_len];
      //idx = (idx+starting[i]) / charset_len;

      // faster version
      // trying to replace mods
      // A % B = A - B * (A/B)
      idx = idx_tmp / charset_len;

      c0[i] = charset[idx_tmp - charset_len * idx];   


    }
  }
  return reset;
}

__device__ void generatePerm(char *c0, int pw_length, char *charset, int charset_len, int idx) {
  // modifies c0 to be a random string perm corresponding to idx in base(charset_len)
	unsigned int len = 0;
	char *c = c0;
  int i;
  int idx_orig = idx;

  for(i = 0; i < 64; i++) {
    if(i >= pw_length) {
      c0[i] = 0;
    } else {
      // kinda dumb once we start doing 0%26; 0/26 over and over again.. but w/e
      // this is essentially a change of base algorithm
      // converts the idx, which is in base10, to base26 if we are you a-z charset. or base62 is a-zA-Z0-9 
      c0[i] = charset[idx % charset_len];
      idx = idx / charset_len;
    }
  }

  //printf("%d = '%s'\n", idx_orig, c0);
}
__device__ inline void md5_prep(char *c0, int pw_length) {
	//unsigned int len = 0;

  char *c = c0 + pw_length;

	//while(*c) {len++; c++;}
	c[0] = 0x80;			// bit 1 after the message


  // this doesn't look right in the bit representation, but maybe that's ok.. http://nsfsecurity.pr.erau.edu/crypto/md5.html might be wrong

  //if(pw_length == len)
  //  printf("YES");

  ((unsigned int*)c0)[14] = pw_length * 8;	// message length in bits

}


//__global__ void md5_kernel(char *charset_d, int charset_len, int pw_length, unsigned char *target_digest_d, int iteration, md5Digest *all_digests_d, int digests_length, md5Plain* returnMD5s)
__global__ void md5_kernel(int *perm_init_index_d, char *charset_d, int charset_len, int pw_length, md5Plain* returnMD5s)
{
  //extern __shared__ md5Node shared_mem[];
  extern __shared__ char shared_mem[];
  char* charset_shared = shared_mem;

  // init shared memory
  for (int i = threadIdx.x; i < charset_len; i += blockDim.x) { 
    if(i < charset_len)
      charset_shared[i] = charset_d[i];
  }
  __syncthreads();

 
  //md5Node *md5Node_arr = (md5Node *) (shared_mem + charset_len + (charset_len%4));
  md5Node *md5Node_arr = (md5Node *) (shared_mem + charset_len + (charset_len & 3));

#ifdef ENABLE_MEMORY
  unsigned char digest[16]; // todo delete?
  char w[64];
#endif
  int hashes_completed;
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  int threads_per_kernel = gridDim.x * blockDim.x;

  // if we have iterated through all permutations already
  bool reset = false;
  //int max_iters = pow((float)charset_len, (float)pw_length);
  //int starting_idx = iteration * MAX_PERMS_PER_KERNEL;


  //for(hashes_completed = starting_idx; hashes_completed < max_iters && (hashes_completed - starting_idx) < MAX_PERMS_PER_KERNEL; hashes_completed += threads_per_kernel) {
  //  // compute a new idx depending on which iteration of the loop we're on.
  //  int overall_idx = idx + hashes_completed;

  for(hashes_completed = idx; hashes_completed < MAX_PERMS_PER_KERNEL; hashes_completed += threads_per_kernel) {
    // compute a new idx depending on which iteration of the loop we're on.
    int overall_idx = hashes_completed;

    if(generatePermStarting(md5Node_arr[threadIdx.x].w, perm_init_index_d, pw_length, charset_shared, charset_len, overall_idx))
    {
      // returned RESET.  There are no more perms
      break; // for
    } else
    {
#ifdef ENABLE_MEMORY
      // we just got a new perm, calculate the md5 for it!
      // save the original string
      for(int i = 0; i < 64; i++) {
        w[i] = md5Node_arr[threadIdx.x].w[i];
      }
#endif

      
      // prepare the md5 string (adding the '1' and the length
	    md5_prep(md5Node_arr[threadIdx.x].w, pw_length);

      // need to do a shuffle for getw...
      GPUshufflegetw((unsigned int *) &(md5Node_arr[threadIdx.x].w));
      // calculate
      md5_v2((unsigned int *) &md5Node_arr[threadIdx.x].w[0], md5Node_arr[threadIdx.x].state[0], md5Node_arr[threadIdx.x].state[1], md5Node_arr[threadIdx.x].state[2], md5Node_arr[threadIdx.x].state[3]);

#ifdef ENABLE_MEMORY
      // encode the 4-wide unsigned int array to a char array
      Encode(digest, md5Node_arr[threadIdx.x].state, 16);
#endif

#ifdef ENABLE_MEMORY
      for(int i =0; i < 16; i++)
        returnMD5s[overall_idx].digest.d[i] = digest[i];
      for(int i =0; i < MAX_PW; i++)
        returnMD5s[overall_idx].plaintext[i] = w[i];
#endif
    }

    //if(overall_idx < max_iters && (overall_idx-starting_idx) < MAX_PERMS_PER_KERNEL) {
    //  // IF we are still a valid permutation AND we have not done more than the maximum number of perms in a single kernel call

    //  // convert w[] to a permutation
    //  generatePerm(md5Node_arr[threadIdx.x].w, pw_length, charset_shared, charset_len, overall_idx);

    //  // save the original string
    //  for(int i = 0; i < 64; i++) {
    //    w[i] = md5Node_arr[threadIdx.x].w[i];
    //  }
    //  
    //  //
    //  // prepare the md5 string (adding the '1' and the length
	   // md5_prep(md5Node_arr[threadIdx.x].w);

    //  // calculate
    //  md5_v2((unsigned int *) &md5Node_arr[threadIdx.x].w[0], md5Node_arr[threadIdx.x].state[0], md5Node_arr[threadIdx.x].state[1], md5Node_arr[threadIdx.x].state[2], md5Node_arr[threadIdx.x].state[3]);

    //  // encode the 4-wide unsigned int array to a char array
    //  Encode(digest, md5Node_arr[threadIdx.x].state, 16);


    //  for(int i =0; i < 16; i++)
    //    returnMD5s[overall_idx - starting_idx].digest.d[i] = digest[i];
    //  for(int i =0; i < 10; i++)
    //    returnMD5s[overall_idx - starting_idx].plaintext[i] = w[i];

      //if(compareMD5Digest(digest, target_digest_d)) {
      //  printf("\n%d="HEXMD5PATTERN"\n", overall_idx, HEXMD5(digest));
      //}
      //if(compareMD5DigestArray(digest, all_digests_d, digests_length)) {
      //  //printf("\n%s="HEXMD5PATTERN"\n", w, HEXMD5(digest));
      //}
    //}
  } // for
}



__global__ void md5_single(char *string_d, int strlen, unsigned char *target_digest_d)
{
  // run md5 on a single string
  md5Node single_md5Node;

  printf("wetrewrfew\n");
  for(int i = 0; i < 64; i++) {
    if(i < strlen)
      single_md5Node.w[i] = string_d[i];
    else
      single_md5Node.w[i] = 0;
  }
  printf("input=%s\n",single_md5Node.w); 
  // prepare the md5 string (adding the '1' and the length
  md5_prep(single_md5Node.w, strlen);

  // calculate
  md5_v2((unsigned int *) &single_md5Node.w[0], single_md5Node.state[0], single_md5Node.state[1], single_md5Node.state[2], single_md5Node.state[3]);

  // encode the 4-wide unsigned int array to a char array
  Encode(target_digest_d, single_md5Node.state, 16);

  //// print out the hash!
  printf(""HEXMD5PATTERN"\n", HEXMD5(target_digest_d));
}

//double diffclock(clock_t clock1,clock_t clock2)
//{
//	double diffticks=clock1-clock2;
//	double diffms=(diffticks*1000)/CLOCKS_PER_SEC;
//	return diffms;
//} 



void MD5StringCuda_pre(int shared_mem_block, char* &charset_h, char* &charset_d, md5Plain* &returnMd5s_h, md5Plain* &returnMd5s_d, int* &perm_init_index_d) {
  // Choose which GPU to run on, change this on a multi-GPU system.
  checkCudaErrors(hipSetDevice(0));
  // every thread will just calc the same md5 hash

  // TODO RE-ENABLE
  //dim3 dimGrid(2, 1, 1);
  //dim3 dimBlock(128, 1, 1);

  // copy charset to GPU
  checkCudaErrors(hipMalloc(&charset_d, strlen(charset_h) * sizeof(char)));
  checkCudaErrors(hipMemcpy(charset_d, charset_h, strlen(charset_h) * sizeof(char), hipMemcpyHostToDevice));

  // malloc the return array of hashes.
  printf("allocating %d bytes\n", MAX_PERMS_PER_KERNEL * sizeof(md5Plain));
#ifdef ENABLE_MEMORY
    checkCudaErrors(hipMalloc(&returnMd5s_d, MAX_PERMS_PER_KERNEL * sizeof(md5Plain)));
#else
    checkCudaErrors(hipMalloc(&returnMd5s_d, 1 * sizeof(md5Plain)));
#endif

  // malloc starting permutation array
  checkCudaErrors(hipMalloc(&perm_init_index_d, MAX_PW * sizeof(int)));
}

bool MD5StringCuda_kernel(dim3 dimGrid, dim3 dimBlock, int shared_mem_block, int *perm_init_index_h, int *perm_init_index_d, char *charset_d, int charset_len, int pw_length, md5Plain *returnMd5s_h, md5Plain *returnMd5s_d) {
  bool error = false;
  // copy the init perm to device
  error = checkCudaErrors(hipMemcpy(perm_init_index_d, perm_init_index_h, MAX_PW * sizeof(int), hipMemcpyHostToDevice));
  if(error)
    return error;

  // invoke kernel
  md5_kernel<<<dimGrid, dimBlock, shared_mem_block>>>(perm_init_index_d, charset_d, charset_len, pw_length, returnMd5s_d);
  error = checkCudaErrors(hipDeviceSynchronize());
  if(error)
    return error;

  // copy the md5s calculated back to the host
#ifdef ENABLE_MEMORY
  checkCudaErrors(hipMemcpy(returnMd5s_h, returnMd5s_d, MAX_PERMS_PER_KERNEL * sizeof(md5Plain), hipMemcpyDeviceToHost));
#endif
}

//void MD5StringCuda(dim3 dimGrid, dim3 dimBlock, char *charset, md5Digest *all_digests_h, int digests_length) {
//  // Choose which GPU to run on, change this on a multi-GPU system.
//  checkCudaErrors(hipSetDevice(0));
//
//
//  // every thread will just calc the same md5 hash
//
//  // TODO RE-ENABLE
//  //dim3 dimGrid(2, 1, 1);
//  //dim3 dimBlock(128, 1, 1);
//
//  char* target_plaintext_h = "abccd";
//  char* target_plaintext_d;
//  unsigned char* target_digest_d;
//  unsigned char target_digest_h[16];
//  checkCudaErrors(hipMalloc(&target_plaintext_d, strlen(target_plaintext_h) * sizeof(char)));
//  checkCudaErrors(hipMemcpy(target_plaintext_d, target_plaintext_h, strlen(target_plaintext_h) * sizeof(char), hipMemcpyHostToDevice));
//  checkCudaErrors(hipMalloc(&target_digest_d, 16 * sizeof(char)));
//
//  // --------
//  md5_single<<<1, 1>>>(target_plaintext_d, strlen(target_plaintext_h), target_digest_d);
//  // --------
//  checkCudaErrors(hipDeviceSynchronize());
//  checkCudaErrors(hipMemcpy(target_digest_h, target_digest_d, 16 * sizeof(char), hipMemcpyDeviceToHost));
//  checkCudaErrors(hipFree(target_plaintext_d));
//  printf("Brute forcing:\n");
//  printf("Hash('%s') == "HEXMD5PATTERN"\n", target_plaintext_h, HEXMD5(target_digest_h));
//
//
//  md5Digest *all_digests_d;
//  printf("length of md5Digest = %d\n", digests_length * sizeof(md5Digest));
//  checkCudaErrors(hipMalloc(&all_digests_d, digests_length * sizeof(md5Digest)));
//  checkCudaErrors(hipMemcpy(all_digests_d, all_digests_h, digests_length * sizeof(md5Digest), hipMemcpyHostToDevice));
//
//
//
//  // Launch a kernel on the GPU with one thread for each element.
//  // sharedmem is 64 bytes for each thread
//  char* charset_d;
//  checkCudaErrors(hipMemcpy(charset_d, charset, strlen(charset) * sizeof(char), hipMemcpyHostToDevice));
//
//  printf("strlen of chraset = %d\n", strlen(charset));
//  // We need 2x ptrs to shared_mem in the kernel. The 2nd array needs to be word-aligned, so we pad the charset
//  int charset_padding = strlen(charset) % 4;
//  int shared_mem_block = dimBlock.x * sizeof(md5Node) + (strlen(charset)+charset_padding) * sizeof(char);
//  printf("shared mem allocated = %d bytes\n", shared_mem_block);
//
//  // put charset in global mem TODO?
//  checkCudaErrors(hipMalloc(&charset_d, strlen(charset) * sizeof(char)));
//  checkCudaErrors(hipMemcpy(charset_d, charset, strlen(charset) * sizeof(char), hipMemcpyHostToDevice));
//
//  // malloc the return array of hashes.
//  md5Plain* returnMd5s_d;
//  md5Plain* returnMd5s_h = new md5Plain[MAX_PERMS_PER_KERNEL];
//  checkCudaErrors(hipMalloc(&returnMd5s_d, MAX_PERMS_PER_KERNEL * sizeof(md5Plain)));
//
//  clock_t begin=clock();
//
//  for(int pw_length = 1; pw_length < 2; pw_length++) {
//    printf("Launching kernel for permutations of length %d ", pw_length);
//    int max_iters = pow((float)strlen(charset), (float)pw_length);
//    for(int j = 0; j <= (max_iters / MAX_PERMS_PER_KERNEL); j++) {
//      printf(".");
//      md5_kernel<<<dimGrid, dimBlock, shared_mem_block>>>(charset_d, strlen(charset), pw_length, target_digest_d, j, all_digests_d, digests_length, returnMd5s_d);
//      checkCudaErrors(hipDeviceSynchronize());
//      checkCudaErrors(hipMemcpy(returnMd5s_h, returnMd5s_d, MAX_PERMS_PER_KERNEL * sizeof(md5Plain), hipMemcpyDeviceToHost));
//
//      cpuCheckMD5Intersection(returnMd5s_h, strlen(charset), pw_length, all_digests_h, digests_length);
//    }
//    printf(" done\n");
//  }
//  clock_t end=clock();
//  printf("Time Elapsed: %lf secs\n", double(diffclock(end, begin)/1000.0));
//
//
//  printf("returned\n");
//  
//
//  return;
//}